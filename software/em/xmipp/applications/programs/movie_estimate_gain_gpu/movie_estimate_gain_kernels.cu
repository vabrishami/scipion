
#include <hip/hip_runtime.h>
#define TILE_DIM 32
#define TILE_DIMH (TILE_DIM/2)
#define TILE_DIM2 (TILE_DIM*2)
#define BLOCK_ROWS 8
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


#define FILL_GPU(d_array, value, Xdim, Ydim) { \
	block.x=floor(((Xdim)+(TILE_DIM-1))/TILE_DIM); block.y=floor(((Ydim)+TILE_DIMH-1)/(TILE_DIMH)); \
	thread.x=TILE_DIM; thread.y=TILE_DIMH; \
	Kfill<<< block, thread >>>((d_array), (value), (Xdim), (Ydim));\
	hipDeviceSynchronize(); \
	gpuErrchk(hipGetLastError()); \
	}

// transpose output (note that the input has Ydim columns!!!)
#define TRANSPOSE_GPU(d_source, d_dest, Xdim, Ydim) { \
	block.x=floor(((Xdim)+(TILE_DIM-1))/TILE_DIM); block.y=floor(((Ydim)+(TILE_DIM-1))/TILE_DIM); \
	thread.x=TILE_DIM; thread.y=BLOCK_ROWS; \
	Ktranspose<<< block, thread >>>((d_source), (d_dest), (Xdim), (Ydim)); \
	hipDeviceSynchronize(); \
	gpuErrchk(hipGetLastError()); \
	}




inline void gpuAssert(hipError_t code, const char *file, int line,
			 bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/****
 sumall - sum no_imgs images in matrix sumObjs, starting from firstImg
****/
__global__
void Ksumall( float* sumObjs, int** array_Img, int no_imgs, int Xdim, int Ydim){

   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int offset = x+y*Xdim;
   int px_img = Xdim*Ydim;
  
   if ((x<Xdim)&&(y<Ydim)){
//   	printf("no_imgs=%d, (%d, %d) th=(%d,%d) blk=(%d,%d)\n", no_imgs, x, y, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
	double sum=0;
	int* ptr;
	for (int img=0; img<no_imgs; img++){
	    	ptr=array_Img[img];
		sum+= (double)ptr[offset];
	}
        sum*=2;
        sumObjs[x+y*Xdim]=(float)sum;
//	printf("data %f\n", sumObjs[x+y+y*Xdim]);

   } //end-if Xdim, Ydim
}//sumall

/****
 mult - c=a+b
****/
__global__
void Kmult( int* a, float* b, int* c, int Xdim, int Ydim){

   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int offset = x+y*Xdim;

   if ((x<Xdim)&&(y<Ydim))
	c[offset]=int((double)a[offset]*b[offset]);
}//mult

/****
 fill - element=value
****/
template<typename T>
__global__
void Kfill( T* a, T value, int Xdim, int Ydim){

   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
   int offset = x+y*Xdim;

   if ((x<Xdim)&&(y<Ydim))
	a[offset]=value;
}//fill



/****
 smooth1: first kernel of constructSmoothHistogramColumn/Row
          if used for constructSmoothHistogramRow, rowH must be transposed
          before calling the kernel, and smooth after calling it
          Each thread computes a semicolumn

          For constructSmoothHistogramsByRow, the colrow must be first transposed
****/

/*********** IT CAN BE OPTIMIZED ************/
/*********** adding more threads in the Y dim ************/
__global__ void Ksmooth1(float *smooth, int *colrow, const float *listOfWeights, int width, int Xdim, int Ydim)
{
   int x = blockIdx.x*TILE_DIM2 + threadIdx.x;
   int y = blockIdx.y*TILE_DIM2 + threadIdx.y;

   if (x<Xdim){
	   double sumWeightsC = 0;
	   int tmp=0;
	   for(int k = -width; k<=width; ++k){
		if (x+k<0 || x+k>=Xdim)
			continue;
		tmp++;
	// DIRECT_A2D_ELEM(smoothColumnH,i,j) += actualWeightC * DIRECT_A2D_ELEM(columnH,i,j+k);
			float actualWeightC = listOfWeights[abs(k)];
		sumWeightsC += actualWeightC;
		for (size_t i=0; i<TILE_DIM2; ++i){
			if ((y+i)<Ydim){
				smooth[x+(y+i)*Xdim] += actualWeightC * (colrow[(x+k)+(y+i)*Xdim]);
			}	
		}	
	  }	
	  if ((width==0)&&(sumWeightsC==0)){
		printf("tmp=%d sumWeightsC=%f width=%d\n", tmp, sumWeightsC, width);	
  		printf("BLK(%d,%d), TH(%d,%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
	  }
	 double iSumWeightsC=1./sumWeightsC;
	  for (size_t i=0; i<TILE_DIM2; ++i)
		if ((y+i)<Ydim)
			smooth[x+(y+i)*Xdim] *= (float)iSumWeightsC;
   }//end-if(x<=Xdim) 
}

/****
 Ksmooth2: average of rows into the first row
          There is a thread per column. A thread averages a whole column into the first element
          A thread copies all the values of the first element into the rest of the elements of the column
 	  (Not the best implementation but easy to code)
          For constructSmoothHistogramsByCol, the smooth matrix must be first transposed

****/
/*********** IT CAN BE OPTIMIZED ************/
/*********** adding more threads in the Y dim ************/
__global__ void Ksmooth2(float *smooth, int Xdim, int Ydim)
{
   int x = blockIdx.x*TILE_DIM2 + threadIdx.x;

   if (x<Xdim){
	   double sum = 0.;
 	   for (size_t y=0; y<Ydim; ++y){
		sum += smooth[x+y*Xdim];
	  }	

	  sum /= Ydim;
 	   for (size_t y=0; y<Ydim; ++y){ 
		smooth[x+y*Xdim]=(float)sum;
	  }	
   }//end-if(x<=Xdim) 
}

/****
K transformGray: each thread deals with a whole column
      1. read value from Iframe - pixval
      2. look for position of first value bigger than pixval - pos
      3. write in IframeTransformedColRow smoothColRos(pos-1) 

****/
__global__ void KtransformGray(const int *Iframe, const int *colrowH, int *IframeTransformColRow, const float* smoothColRow, int Xdim, int Ydim)
{
   int x = blockIdx.x*TILE_DIM2 + threadIdx.x;
 
   for (size_t y0=0; y0<Ydim; ++y0){
	   int pixval = Iframe[x+y0*Xdim];
		// upperbounds
	   size_t y1;
   	   for (y1=0; y1<Ydim; ++y1){
		if (colrowH[x+y1*Xdim]>pixval)
			break;
	   } 
           if (y1==Ydim)
		y1--;
	__syncthreads();   
	IframeTransformColRow[x+y0*Xdim]=(int)smoothColRow[x+y1*Xdim];
   }//end-for-y0	

}



// Kernel to transpose a matrix

// The kernel assumes that each block deals with a 32x32 tile
// each block has 32x8 threads 
// IMPORTANT: The input has Xdim columns X Ydim rows
//            The output has Ydim columns X Xdim rows
template <typename T>
__global__ void Ktranspose(T *odata, const T *idata, int Xdim, int Ydim)
{
__shared__ float tile[TILE_DIM+1][TILE_DIM];
//__shared__ float tile[TILE_DIM][TILE_DIM];
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

  if (x<Xdim)
	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		if (y+j<Ydim)
		     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*Xdim + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  if (x<Ydim)
	  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		if (y+j<Xdim)
		     odata[(y+j)*Ydim + x] = tile[threadIdx.x][threadIdx.y + j];
}



